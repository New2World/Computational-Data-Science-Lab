#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <hiprand/hiprand_kernel.h>

#define QUE_LEN 2000
#define QUE_ST(x) (x * QUE_LEN)
#define QUE_ED(x) (x * QUE_LEN + QUE_LEN)

#define RAND_FACTOR 1e9+7

#define MAX_OUT 2000
#define MAX_NODE 10000

#define THREAD 5
#define CONSTANT_PROBABILITY 0.1

int _adjMat[MAX_NODE][MAX_OUT];

int readGraph(const char* filePath,
              int* adjCount,
              int* adjList,
              int& nodes,
              int& outdegree){
    int s, t, count = 0;
    FILE* fd = fopen(filePath, "r");
    if(fd == NULL)
        return -1;
    memset(adjCount, 0, sizeof(int) * MAX_NODE);
    while(!feof(fd)){
        fscanf(fd, "%d %d", &s, &t);
        _adjMat[s - 1][adjCount[s]++] = t - 1;
        count++;
        nodes = nodes > s ? (nodes > t ? nodes : t) : (s > t ? s : t);
    }
    fclose(fd);
    int ptr = 0;
    for(int i = 0;i < nodes;i++){
        memcpy(adjList + ptr, _adjMat[i], sizeof(int) * adjCount[i + 1]);
        ptr += adjCount[i + 1];
        outdegree = outdegree > adjCount[i + 1] ? outdegree : adjCount[i + 1];
        adjCount[i + 1] += adjCount[i];
    }
    return count;
}

__device__ int getIndex(){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int line = blockDim.x * gridDim.x;
    return row * line + col;
}

__device__ int findVertice(int* nodeSet, int from, int nodes){
    for(int i = from + 1;i < nodes;i++)
        if(atomicCAS(&nodeSet[i], -1, 0) == -1)
            return i;
    return from;
}

__device__ bool que_isFull(int que_h, int que_t){
    return que_t == que_h;
}

__device__ bool que_isEmpty(int que_h, int que_t, int index){
    return (que_t == que_h + 1) || (que_t == que_h + 1 - QUE_LEN);
}

__device__ void que_clear(int& que_h, int& que_t, int index){
    que_h = QUE_ST(index);
    que_t = que_h + 1;
}

__device__ bool que_enque(int* queue, int que_h, int& que_t, int val, int index){
    if(que_isFull(que_h, que_t))
        return false;
    int tail = que_t - 1;
    if(tail < QUE_ST(index))
        tail += QUE_LEN;
    queue[tail] = val;
    que_t++;
    if(que_t >= QUE_ED(index))
        que_t -= QUE_LEN;
    return true;
}

__device__ int que_deque(int* queue, int& que_h, int que_t, int index){
    int val = -1;
    if(que_isEmpty(que_h, que_t, index))
        return val;
    val = queue[que_h];
    que_h++;
    if(que_h >= QUE_ED(index))
        que_h -= QUE_LEN;
    return val;
}

__device__ bool nd_isVisited(bool* vis, int nd, int index){
    return vis[index * THREAD + nd];
}

__device__ void nd_setVisited(bool* vis, int nd, int index){
    vis[index * THREAD + nd] = true;
}

__global__ void setupRandGenerator(float* randSeed, hiprandState* state){
    int index = getIndex();
    unsigned long seed = (unsigned long)(randSeed[index] * RAND_FACTOR);
    hiprand_init(seed, index, 0, &state[index]);
}

__global__ void bfs(int totalNodes,
                    int* adjCount,
                    int* adjList,
                    int* nodeSet,
                    int* queue,
                    bool* closed,
                    hiprandState* state){
    int index = getIndex();
    int count = 0, node = index;
    int next, prev = -1;
    float randProb;
    int que_h = QUE_ST(index), que_t = que_h + 1;
    hiprandState localState = state[index];
    while(node != prev){
        prev = node;
        que_clear(que_h, que_t, index);
        if(!que_enque(queue, que_h, que_t, node, index));   // in case queue overflow
        nd_setVisited(closed, prev, index);
        while(!que_isEmpty(que_h, que_t, index)){
            node = que_deque(queue, que_h, que_t, index);
            next = adjCount[node];
            while(next < adjCount[node + 1]){
                if(!nd_isVisited(closed, adjList[next], index)){
                    randProb = hiprand_uniform(&localState);
                    if(randProb < CONSTANT_PROBABILITY){
                        if(!que_enque(queue, que_h, que_t, adjList[next], index));
                        nd_setVisited(closed, adjList[next], index);
                        count++;
                    }
                }
                next++;
            }
        }
        if(atomicCAS(nodeSet + prev, 0, count) != 0);   // theoretically impossiable
        node = findVertice(nodeSet, prev, totalNodes);
    }
    state[index] = localState;
}

int h_adjCount[MAX_NODE];
int h_adjList[MAX_NODE * MAX_OUT];
int h_nodeSet[MAX_NODE];

int main(){

    int totalNodes = 0, maxOutDegree = 0;
    int totalEdges = readGraph("../data/test.txt", h_adjCount, h_adjList, totalNodes, maxOutDegree);
    if(totalEdges < 0)
        return 0;
    printf("This graph contains %d nodes connected by %d edges\n", totalNodes, totalEdges);

    bool* d_closed;
    int* d_queue, *d_nodeSet;
    int* d_adjList, *d_adjCount;
    float* d_randSeed;
    hiprandState* d_randState;

    dim3 gridSize(1,1), blockSize(1,THREAD);

    hiprandGenerator_t curandGenerator;
    hipMalloc((void**)&d_randSeed, sizeof(float) * THREAD);
    hipMalloc((void**)&d_randState, sizeof(hiprandState) * THREAD);
    hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curandGenerator, time(NULL));
    hiprandGenerateUniform(curandGenerator, d_randSeed, THREAD);
    setupRandGenerator<<<gridSize,blockSize>>>(d_randSeed, d_randState);

    hipMalloc((void**)&d_closed, sizeof(bool) * THREAD * totalNodes);
    hipMalloc((void**)&d_queue, sizeof(int) * THREAD * QUE_LEN);    // compress?
    hipMalloc((void**)&d_nodeSet, sizeof(int) * totalNodes);
    hipMalloc((void**)&d_adjList, sizeof(int) * totalEdges);
    hipMalloc((void**)&d_adjCount, sizeof(int) * (totalNodes + 1));

    hipMemset(d_closed, false, sizeof(bool) * THREAD * totalNodes);
    hipMemcpy(d_adjList, h_adjList, sizeof(int) * totalEdges, hipMemcpyHostToDevice);
    hipMemcpy(d_adjCount,
               h_adjCount,
               sizeof(int) * (totalNodes + 1),
               hipMemcpyHostToDevice);

    bfs<<<gridSize,blockSize>>>(totalNodes,
                                d_adjCount,
                                d_adjList,
                                d_nodeSet,
                                d_queue,
                                d_closed,
                                d_randState);

    hipMemcpy(h_nodeSet, d_nodeSet, sizeof(int) * totalNodes, hipMemcpyDeviceToHost);

    float avg = 0.;
    for(int i = 0;i < totalNodes;i++){
        printf("influence of node %d: %d\n", i, h_nodeSet[i]);
        avg += 1. * h_nodeSet[i];
    }
    avg /= totalNodes;
    printf("average influence %.2f nodes\n", avg);

    hipFree(d_randSeed);
    hipFree(d_randState);
    hipFree(d_closed);
    hipFree(d_queue);
    hipFree(d_nodeSet);
    hipFree(d_adjList);
    hipFree(d_adjCount);

    return 0;
}
