#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <getopt.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "cuqueue.cuh"

#define RAND_FACTOR 1e9+7

#define MAX_OUT 2000
#define MAX_NODE 10000
#define THREAD 1024

float CONSTANT_PROBABILITY = 0.01;
int _adjMat[MAX_NODE][MAX_OUT];

// read graph information from files and generate a adjacent list for representation
int readGraph(const char* filePath,
              int* adjCount,
              int* adjList,
              int& nodes,
              int& outdegree){
    int s, t, count = 0;
    FILE* fd = fopen(filePath, "r");
    if(fd == NULL)
        return -1;
    memset(adjCount, 0, sizeof(int) * MAX_NODE);
    while(!feof(fd)){
        fscanf(fd, "%d %d", &s, &t);
        _adjMat[s - 1][adjCount[s]++] = t - 1;
        count++;
        nodes = nodes > s ? (nodes > t ? nodes : t) : (s > t ? s : t);
    }
    fclose(fd);
    int ptr = 0;
    for(int i = 0;i < nodes;i++){
        memcpy(adjList + ptr, _adjMat[i], sizeof(int) * adjCount[i + 1]);
        ptr += adjCount[i + 1];
        outdegree = outdegree > adjCount[i + 1] ? outdegree : adjCount[i + 1];
        adjCount[i + 1] += adjCount[i];
    }
    return count;
}

// get thread index
__device__ int getIndex(){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int line = blockDim.x * gridDim.x;
    return row * line + col;
}

// find next unused node
__device__ int findVertice(int* nodeSet, int from, int nodes){
    for(int i = from + 1;i < nodes;i++)
        if(atomicCAS(&nodeSet[i], -1, 0) == -1)
            return i;
    return from;
}

// judge if node 'nd' is visited once
__device__ bool nd_isVisited(bool* vis, int nd, int index){
    return vis[index * THREAD + nd];
}

// set node 'nd' visited
__device__ void nd_setVisited(bool* vis, int nd, int index){
    vis[index * THREAD + nd] = true;
}

// initialize random seeds for each thread
__global__ void setupRandGenerator(float* randSeed, hiprandState* state){
    int index = getIndex();
    unsigned long seed = (unsigned long)(randSeed[index] * RAND_FACTOR);
    hiprand_init(seed, index, 0, &state[index]);
}

// BFS kernel function in each thread
__global__ void bfs(int totalNodes,
                    int* adjCount,
                    int* adjList,
                    int* nodeSet,
                    int* queue,
                    bool* closed,
                    hiprandState* state,
                    float constProb){
    int index = getIndex();
    int count = 0, node = index;
    int que_h, que_t;
    int next, prev = -1;
    float randProb;
    hiprandState localState = state[index];
    while(node != prev){
        prev = node;
        que_init(que_h, que_t, index);
        if(!que_enque(queue, que_h, que_t, prev, index));   // in case queue overflow
        nd_setVisited(closed, prev, index);
        while(!que_isEmpty(que_h, que_t, index)){
            node = que_deque(queue, que_h, que_t, index);
            next = adjCount[node];
            while(next < adjCount[node + 1]){
                if(!nd_isVisited(closed, adjList[next], index)){
                    randProb = hiprand_uniform(&localState);
                    if(randProb < constProb){
                        if(!que_enque(queue, que_h, que_t, adjList[next], index));
                        nd_setVisited(closed, adjList[next], index);
                        count++;
                    }
                }
                next++;
            }
        }
        if(atomicCAS(nodeSet + prev, 0, count) != 0);   // theoretically impossiable
        node = findVertice(nodeSet, prev, totalNodes);
    }
    state[index] = localState;
}

// global variables
int h_adjCount[MAX_NODE];
int h_adjList[MAX_NODE * MAX_OUT];
int h_nodeSet[MAX_NODE];

// for argument parsing
char short_options[] = "p::o";
struct option long_options[]{
    {"probability", optional_argument, 0, 'p'},
    {"output", no_argument, 0, 'o'}
};

int main(int argc, char** argv){
    // argument parsing
    char ch;
    while((ch = getopt_long(argc, argv, short_options, long_options, NULL)) != -1){
        switch(ch){
        case 'p':
            CONSTANT_PROBABILITY = atof(optarg);
            break;
        case 'o':
            freopen("../outputs/dynamicOutput.txt", "a", stdout);
            break;
        }
    }

    // read graph from file
    int totalNodes = 0, maxOutDegree = 0;
    int totalEdges = readGraph("../data/wiki.txt", h_adjCount, h_adjList, totalNodes, maxOutDegree);
    if(totalEdges < 0)
        return 0;
    printf("========= NEW RUN\n");
    printf("This graph contains %d nodes connected by %d edges\n", totalNodes, totalEdges);
    printf("Set constant probability: %.2f\n", CONSTANT_PROBABILITY);

    // addresses for GPU memory addresses storage
    bool* d_closed;
    int* d_queue, *d_nodeSet;
    int* d_adjList, *d_adjCount;
    float* d_randSeed;
    float gpu_runtime;

    hiprandState* d_randState;
    hipEvent_t start, stop;

    // define GPU thread layout
    dim3 gridSize(1,1), blockSize(32,32);

    // generate random numbers for each thread as random seeds
    hiprandGenerator_t curandGenerator;
    hipMalloc((void**)&d_randSeed, sizeof(float) * THREAD);
    hipMalloc((void**)&d_randState, sizeof(hiprandState) * THREAD);
    hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curandGenerator, time(NULL));
    hiprandGenerateUniform(curandGenerator, d_randSeed, THREAD);
    setupRandGenerator<<<gridSize,blockSize>>>(d_randSeed, d_randState);

    // cuda memory allocation
    hipMalloc((void**)&d_closed, sizeof(bool) * THREAD * totalNodes);
    hipMalloc((void**)&d_queue, sizeof(int) * THREAD * QUE_LEN);    // compress?
    hipMalloc((void**)&d_nodeSet, sizeof(int) * totalNodes);
    hipMalloc((void**)&d_adjList, sizeof(int) * totalEdges);
    hipMalloc((void**)&d_adjCount, sizeof(int) * (totalNodes + 1));

    hipMemset(d_closed, false, sizeof(bool) * THREAD * totalNodes);
    hipMemcpy(d_adjList, h_adjList, sizeof(int) * totalEdges, hipMemcpyHostToDevice);
    hipMemcpy(d_adjCount,
               h_adjCount,
               sizeof(int) * (totalNodes + 1),
               hipMemcpyHostToDevice);

    // elapsed time record
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // launch the kernel
    bfs<<<gridSize,blockSize>>>(totalNodes,
                                d_adjCount,
                                d_adjList,
                                d_nodeSet,
                                d_queue,
                                d_closed,
                                d_randState,
                                CONSTANT_PROBABILITY);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_runtime, start, stop);

    hipMemcpy(h_nodeSet, d_nodeSet, sizeof(int) * totalNodes, hipMemcpyDeviceToHost);

    // statistics
    for(int i = 0;i < totalNodes;i++)
        if(h_nodeSet[i] > 0)
            printf("influence of node %d: %d\n", i, h_nodeSet[i]);
    printf("========= GPU ELAPSED TIME: %f ms\n\n", gpu_runtime);

    // cuda memory free
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_randSeed);
    hipFree(d_randState);
    hipFree(d_closed);
    hipFree(d_queue);
    hipFree(d_nodeSet);
    hipFree(d_adjList);
    hipFree(d_adjCount);

    return 0;
}
